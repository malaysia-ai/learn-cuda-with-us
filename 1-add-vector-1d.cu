#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 1000000;
    size_t size = n * sizeof(int);
    
    int *h_a, *h_b, *h_c;
    
    int *d_a, *d_b, *d_c;
    
    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);
    
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < n; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            printf("Error: %d + %d != %d\n", h_a[i], h_b[i], h_c[i]);
            break;
        }
    }
    
    printf("Vector addition completed successfully.\n");
    
    free(h_a); free(h_b); free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    return 0;
}