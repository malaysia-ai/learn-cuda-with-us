#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *a, int *b, int *c, int columns) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int i = row * columns + col;
    c[i] = a[i] + b[i];
    
}

int main() {
    int rows = 1000;
    int columns = 1000;
    size_t size = rows * columns * sizeof(int);
    
    int *h_a, *h_b, *h_c;
    
    int *d_a, *d_b, *d_c;
    
    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);
    
    for (int i = 0; i < rows * columns; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
    // 16 * 16 = 256
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((columns + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);
    vectorAdd<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, columns);
    
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < rows * columns; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            printf("Error: %d + %d != %d\n", h_a[i], h_b[i], h_c[i]);
            break;
        }
    }
    
    printf("Vector addition completed successfully.\n");
    
    free(h_a); free(h_b); free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    return 0;
}